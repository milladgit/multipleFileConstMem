#include <stdio.h>
#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

__device__ extern double access_const_mem(int index);
extern void init_const_mem(double initValue);


__global__
void const_kernel() {
	for(int i=0;i<10;i++) {
		printf("%d: %.3f\n", i, access_const_mem(i));
	}
}

int main() {
	init_const_mem(23);

	const_kernel<<<1, 1>>>();

	hipDeviceSynchronize();

	return 0;
}
