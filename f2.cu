
#include <hip/hip_runtime.h>

#define N 100


extern __constant__ double buffer[];


void init_const_mem(double initValue) {
	double *a = (double*) malloc(N * sizeof(double));
	for(int i=0;i<N;i++) {
		a[i] = initValue * i;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(buffer), a, N * sizeof(double));
	free(a);
}


__device__
double access_const_mem(int index) {
	return buffer[index];
}



